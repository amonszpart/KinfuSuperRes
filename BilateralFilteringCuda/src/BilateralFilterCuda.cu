#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <hip/hip_vector_types.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>       // CUDA device initialization helper functions

/*
    Perform a simple bilateral filter.

    Bilateral filter is a nonlinear filter that is a mixture of range
    filter and domain filter, the previous one preserves crisp edges and
    the latter one filters noise. The intensity value at each pixel in
    an image is replaced by a weighted average of intensity values from
    nearby pixels.

    The weight factor is calculated by the product of domain filter
    component(using the gaussian distribution as a spatial distance) as
    well as range filter component(Euclidean distance between center pixel
    and the current neighbor pixel). Because this process is nonlinear,
    the sample just uses a simple pixel by pixel step.

    Texture fetches automatically clamp to edge of image. 1D gaussian array
    is mapped to a 1D texture instead of using shared memory, which may
    cause severe bank conflict.

    Threads are y-pass(column-pass), because the output is coalesced.

    Parameters
    od - pointer to output data in global memory
    d_f - pointer to the 1D gaussian array
    e_d - euclidean delta
    w  - image width
    h  - image height
    r  - filter radius
*/

//// GLOBALS

__constant__ float cGaussian[64];   //gaussian array in device side

typedef texture<uchar4, 2, hipReadModeNormalizedFloat> TextureU4f;
TextureU4f rgbaTex;

typedef texture<uchar4, 2, hipReadModeNormalizedFloat> TextureU4f;
TextureU4f guideTex;

typedef texture<float, 2, hipReadModeElementType> TextureU1f;
TextureU1f depthTex;

//// HELPERS

// Euclidean Distance (x, y, d) = exp((|x - y| / d)^2 / 2)
__device__ float euclideanLen(float4 a, float4 b, float d)
{

    float mod = (b.x - a.x) * (b.x - a.x) +
                (b.y - a.y) * (b.y - a.y) +
                (b.z - a.z) * (b.z - a.z);

    return __expf(-mod / (2.f * d * d));
}

__device__ float euclideanLen( float a, float b, float d )
{

    float diff = (b - a);

    return __expf( -(diff*diff) / (2.f * d * d) );
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(fabs(rgba.x));   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(fabs(rgba.y));
    rgba.z = __saturatef(fabs(rgba.z));
    rgba.w = __saturatef(fabs(rgba.w));
    return (uint(rgba.w * 255.0f) << 24) | (uint(rgba.z * 255.0f) << 16) | (uint(rgba.y * 255.0f) << 8) | uint(rgba.x * 255.0f);
}

__device__ float4 rgbaIntToFloat(uint c)
{
    float4 rgba;
    rgba.x = (c & 0xff) * 0.003921568627f;       //  /255.0f;
    rgba.y = ((c>>8) & 0xff) * 0.003921568627f;  //  /255.0f;
    rgba.z = ((c>>16) & 0xff) * 0.003921568627f; //  /255.0f;
    rgba.w = ((c>>24) & 0xff) * 0.003921568627f; //  /255.0f;
    return rgba;
}

//// PRECOMPUTATION

/*
    Because a 2D gaussian mask is symmetry in row and column,
    here only generate a 1D mask, and use the product by row
    and column index later.

    1D gaussian distribution :
        g(x, d) -- C * exp(-x^2/d^2), C is a constant amplifier

    parameters:
    og - output gaussian array in global memory
    delta - the 2nd parameter 'd' in the above function
    radius - half of the filter size
             (total filter size = 2 * radius + 1)
*/
extern "C"
void updateGaussian(float delta, int radius)
{
    float  fGaussian[64];

    for (int i = 0; i < 2*radius + 1; ++i)
    {
        float x = i-radius;
        fGaussian[i] = expf(-(x*x) / (2*delta*delta));
    }

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cGaussian), fGaussian, sizeof(float)*(2*radius+1)));
}

//// Bilateral RGBA (8UC4)

__global__ void
d_bilateral_filterRGBA( uint *od, int w, int h, float e_d, int r )
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= w || y >= h)
    {
        return;
    }

    float sum = 0.0f;
    float factor;
    float4 t = {0.f, 0.f, 0.f, 0.f};
    float4 center = tex2D(rgbaTex, x, y);

    for (int i = -r; i <= r; ++i)
    {
        for (int j = -r; j <= r; ++j)
        {
            float4 curPix = tex2D(rgbaTex, x + j, y + i);
            if ( curPix.x == 0 )
                continue;
            factor = cGaussian[i + r] * cGaussian[j + r] *     //domain factor
                     euclideanLen(curPix, center, e_d);             //range factor

            t += factor * curPix;
            sum += factor;
        }
    }

    od[y * w + x] = rgbaFloatToInt(t/sum);
}

/*
    Perform 2D bilateral filter on image using CUDA

    Parameters:
    d_dest - pointer to destination image in device memory
    width  - image width
    height - image height
    e_d    - euclidean delta
    radius - filter radius
    iterations - number of iterations
*/
extern "C"
double bilateralFilterRGBA(uint *dDest,
                           int width, int height,
                           float e_d, int radius, int iterations,
                           StopWatchInterface *timer,
                           uint* dImage, uint* dTemp, uint pitch )
{
    // var for kernel computation timing
    double dKernelTime;

    // Bind the array to the texture
    hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
    size_t offset = -1;
    checkCudaErrors( hipBindTexture2D(&offset, rgbaTex, dImage, desc, width, height, pitch) );
    if ( offset > 0 )
    {
        std::cerr << "hipBindTexture2D returne non-zero offset!!!" << std::endl;
    }

    for (int i=0; i<iterations; i++)
    {
        // sync host and start kernel computation timer
        dKernelTime = 0.0;
        checkCudaErrors(hipDeviceSynchronize());
        sdkResetTimer(&timer);

        dim3 gridSize((width + 16 - 1) / 16, (height + 16 - 1) / 16);
        dim3 blockSize(16, 16);
        d_bilateral_filterRGBA<<< gridSize, blockSize>>>(
                                                           dDest, width, height, e_d, radius );

        // sync host and stop computation timer
        checkCudaErrors(hipDeviceSynchronize());
        dKernelTime += sdkGetTimerValue(&timer);

        if (iterations > 1)
        {
            // copy result back from global memory to array
            checkCudaErrors(hipMemcpy2D(dTemp, pitch, dDest, sizeof(int)*width,
                                         sizeof(int)*width, height, hipMemcpyDeviceToDevice));
            checkCudaErrors(hipBindTexture2D(0, rgbaTex, dTemp, desc, width, height, pitch));
        }
    }

    return ((dKernelTime/1000.)/(double)iterations);
}

//// Bilateral 32FC1

__global__ void
d_bilateral_filterF( float *od, int w, int h, float e_d, int r )
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= w || y >= h)
    {
        return;
    }

    float sum = 0.0f;
    float factor;
    float t = 0.f;
    float center = tex2D( depthTex, x, y );

    for ( int i = -r; i <= r; ++i )
    {
        for ( int j = -r; j <= r; ++j )
        {
            float curPix = tex2D(depthTex, x + j, y + i);

            if ( curPix == 0.f ) // skip, if empty
                continue;

            factor = cGaussian[i + r] * cGaussian[j + r] *     // domain factor
                     euclideanLen(curPix, center, e_d);        // range factor

            t   += factor * curPix;
            sum += factor;
        }
    }

    // output
    od[y * w + x] = t / sum;
}

extern "C"
double bilateralFilterF( float *dDest,
                         int width, int height,
                         float e_d, int radius, int iterations,
                         StopWatchInterface *timer,
                         float* dImage, float* dTemp, uint pitch )
{
    // var for kernel computation timing
    double dKernelTime;

    // Bind the array to the texture
    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    size_t offset = -1;
    checkCudaErrors( hipBindTexture2D(&offset, depthTex, dImage, desc, width, height, pitch) );
    if ( offset > 0 )
    {
        std::cerr << "hipBindTexture2D returne non-zero offset!!!" << std::endl;
    }

    for (int i=0; i<iterations; i++)
    {
        // sync host and start kernel computation timer
        dKernelTime = 0.0;
        checkCudaErrors(hipDeviceSynchronize());
        sdkResetTimer(&timer);

        dim3 gridSize((width + 16 - 1) / 16, (height + 16 - 1) / 16);
        dim3 blockSize(16, 16);
        d_bilateral_filterF<<< gridSize, blockSize>>>( dDest, width, height, e_d, radius );

        // sync host and stop computation timer
        checkCudaErrors(hipDeviceSynchronize());
        dKernelTime += sdkGetTimerValue(&timer);

        if (iterations > 1)
        {
            // copy result back from global memory to array
            checkCudaErrors(hipMemcpy2D(dTemp, pitch, dDest, sizeof(int)*width,
                                         sizeof(int)*width, height, hipMemcpyDeviceToDevice));
            checkCudaErrors(hipBindTexture2D(0, rgbaTex, dTemp, desc, width, height, pitch));
        }
    }

    return ((dKernelTime/1000.)/(double)iterations);
}


//// CrossBilateral RGBA (8UC4)

__global__ void
d_cross_bilateral_filterRGBA( uint *od, int w, int h, float e_d, int r )
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= w || y >= h)
    {
        return;
    }

    float sum = 0.0f;
    float factor;
    float4 t = {0.f, 0.f, 0.f, 0.f};
    float4 center = tex2D(guideTex, x, y);

    for (int i = -r; i <= r; ++i)
    {
        for (int j = -r; j <= r; ++j)
        {
            float4 curPix = tex2D( rgbaTex, x + j, y + i );
            float4 guidePix = tex2D( guideTex, x + j, y + i );
            if ( curPix.x == 0 )
                continue;
            factor = cGaussian[i + r] * cGaussian[j + r] *     //domain factor
                     euclideanLen(guidePix, center, e_d);             //range factor

            t += factor * curPix;
            sum += factor;
        }
    }

    od[y * w + x] = rgbaFloatToInt(t/sum);
}

extern "C"
double crossBilateralFilterRGBA( uint *dDest,
                                 uint *dImage, uint *dTemp, uint pitch,
                                 uint *dGuide, uint guidePitch,
                                 int width, int height,
                                 float e_d, int radius, int iterations,
                                 StopWatchInterface *timer
                                 )
{
    // var for kernel computation timing
    double dKernelTime;

    // Bind the array to the texture
    hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
    size_t offset = -1;
    checkCudaErrors( hipBindTexture2D(&offset, rgbaTex, dImage, desc, width, height, pitch) );
    if ( offset > 0 )
    {
        std::cerr << "hipBindTexture2D returne non-zero offset!!!" << std::endl;
    }

    checkCudaErrors( hipBindTexture2D(&offset, guideTex, dGuide, desc, width, height, guidePitch) );
    if ( offset > 0 )
    {
        std::cerr << "hipBindTexture2D returne non-zero offset!!!" << std::endl;
    }


    for (int i=0; i<iterations; i++)
    {
        // sync host and start kernel computation timer
        dKernelTime = 0.0;
        checkCudaErrors(hipDeviceSynchronize());
        sdkResetTimer(&timer);

        dim3 gridSize((width + 16 - 1) / 16, (height + 16 - 1) / 16);
        dim3 blockSize(16, 16);
        d_cross_bilateral_filterRGBA<<< gridSize, blockSize>>>(
                                                                 dDest, width, height, e_d, radius );

        // sync host and stop computation timer
        checkCudaErrors(hipDeviceSynchronize());
        dKernelTime += sdkGetTimerValue(&timer);

        if (iterations > 1)
        {
            // copy result back from global memory to array
            checkCudaErrors(hipMemcpy2D(dTemp, pitch, dDest, sizeof(int)*width,
                                         sizeof(int)*width, height, hipMemcpyDeviceToDevice));
            checkCudaErrors(hipBindTexture2D(0, rgbaTex, dTemp, desc, width, height, pitch));
        }
    }

    return ((dKernelTime/1000.)/(double)iterations);
}


//// CrossBilateral 32FC1

__global__ void
d_cross_bilateral_filterF( float *od, int w, int h, float e_d, int r )
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= w || y >= h)
    {
        return;
    }

    float sum = 0.0f;
    float factor;
    float t = 0.f;
    float4 center = tex2D(guideTex, x, y);

    for ( int i = -r; i <= r; ++i )
    {
        for ( int j = -r; j <= r; ++j )
        {
            float curPix   = tex2D( depthTex, x + j, y + i );
            float4 guidePix = tex2D( guideTex, x + j, y + i );
            if ( curPix == 0.f )
                continue;
            factor = cGaussian[i + r] * cGaussian[j + r] *     //domain factor
                     euclideanLen( guidePix, center, e_d );             //range factor

            t   += factor * curPix;
            sum += factor;
        }
    }

    od[y * w + x] = t / sum;
    //od[y * w + x] = tex2D( depthTex, x , y );
}

extern "C"
double crossBilateralFilterF( float *dDest,
                              float *dImage, float *dTemp, uint pitch,
                              uint *dGuide, uint guidePitch,
                              int width, int height,
                              float e_d, int radius, int iterations,
                              StopWatchInterface *timer
                              )
{
    // var for kernel computation timing
    double dKernelTime;

    // Bind the array to the texture
    hipChannelFormatDesc descF = hipCreateChannelDesc<float>();
    size_t offset = -1;
    checkCudaErrors( hipBindTexture2D(&offset, depthTex, dImage, descF, width, height, pitch) );
    if ( offset > 0 )
    {
        std::cerr << "hipBindTexture2D returne non-zero offset!!!" << std::endl;
    }

    hipChannelFormatDesc descU4 = hipCreateChannelDesc<uchar4>();
    checkCudaErrors( hipBindTexture2D(&offset, guideTex, dGuide, descU4, width, height, guidePitch) );
    if ( offset > 0 )
    {
        std::cerr << "hipBindTexture2D returne non-zero offset!!!" << std::endl;
    }


    for (int i=0; i<iterations; i++)
    {
        // sync host and start kernel computation timer
        dKernelTime = 0.0;
        checkCudaErrors(hipDeviceSynchronize());
        sdkResetTimer(&timer);

        dim3 gridSize((width + 16 - 1) / 16, (height + 16 - 1) / 16);
        dim3 blockSize(16, 16);
        d_cross_bilateral_filterF<<< gridSize, blockSize>>>( dDest, width, height, e_d, radius );

        // sync host and stop computation timer
        checkCudaErrors(hipDeviceSynchronize());
        dKernelTime += sdkGetTimerValue(&timer);

        if (iterations > 1)
        {
            // copy result back from global memory to array
            checkCudaErrors(hipMemcpy2D(dTemp, pitch, dDest, sizeof(int)*width,
                                         sizeof(int)*width, height, hipMemcpyDeviceToDevice));
            checkCudaErrors(hipBindTexture2D(0, rgbaTex, dTemp, descF, width, height, pitch));
        }
    }

    return ((dKernelTime/1000.)/(double)iterations);
}


